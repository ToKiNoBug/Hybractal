#include "hip/hip_runtime.h"

#include "cubractal.h"
#include <hip/hip_complex.h>
#include <hip/hip_runtime.h>
#include <cuda_complex.hpp>
#include <exception>
#include <fmt/format.h>

libHybractal::cubractal_resource::cubractal_resource(size_t __rows,
                                                     size_t __cols)
    : _rows(__rows), _cols(__cols), gpu_age(nullptr), gpu_z(nullptr) {
  const size_t elements = this->size();
  if (elements <= 0) {
    return;
  }
  hipError_t err;
  err = hipMalloc(&this->gpu_age, elements * sizeof(uint16_t));
  if (err != hipSuccess) {
    return;
  }
  err = hipMalloc(&this->gpu_z, elements * sizeof(std::complex<double>));
  if (err != hipSuccess) {
    return;
  }
}

libHybractal::cubractal_resource::~cubractal_resource() {
  if (this->gpu_age != nullptr) {
    hipFree(this->gpu_age);
  }
  if (this->gpu_z != nullptr) {
    hipFree(this->gpu_z);
  }
}

libHybractal::cubractal_resource::cubractal_resource(cubractal_resource &&src)
    : _rows(src._rows), _cols(src.cols()), gpu_age(src.gpu_age),
      gpu_z(src.gpu_z) {
  src.gpu_age = nullptr;
  src.gpu_z = nullptr;
}

libHybractal::cubractal_resource &
libHybractal::cubractal_resource::operator=(cubractal_resource &&src) {
  this->_rows = src._rows;
  this->_cols = src._cols;
  this->gpu_age = src.gpu_age;
  this->gpu_z = src.gpu_z;
  src.gpu_age = nullptr;
  src.gpu_z = nullptr;
  return *this;
}

template <typename flt_t> using cuda_complex_t = complex<flt_t>;

struct cuda_wind {
  cuda_complex_t<double> top_left;
  double r_unit;
  double c_unit;
};

template <typename flt_t>
void __global__ compute_by_block(std::array<int, 2> size_rc, cuda_wind wind,
                                 uint16_t maxit, uint16_t *age_ptr,
                                 hipDoubleComplex *z_ptr) {
  // x+ <=> c+
  // y+ <=> r+

  using cplx_t = cuda_complex_t<flt_t>;
  const int c = threadIdx.x + blockDim.x * blockIdx.x;
  const int r = threadIdx.y + blockDim.y * blockIdx.y;
  assert(r >= 0 && r < size_rc[0]);
  assert(c >= 0 && c < size_rc[1]);

  const cuda_complex_t<double> C_double =
      cuda_complex_t<double>{wind.top_left} +
      cuda_complex_t<double>{wind.c_unit * c, wind.r_unit * r};
  const cplx_t C{flt_t(C_double.real()), flt_t(C_double.imag())};

  cplx_t z{0, 0};

  int age = DECLARE_HYBRACTAL_SEQUENCE(
      HYBRACTAL_SEQUENCE_STR)::compute_age<float_t, cplx_t>(z, C, maxit);
  if (age < 0) {
    age = UINT16_MAX;
  }

  const int global_idx = r * size_rc[1] + c;
  age_ptr[global_idx] = age;
  if (z_ptr != nullptr) {
    z_ptr[global_idx] = {z.real(), z.imag()};
  }
  // hipErrorAssert(r >= 0 && r < size_rc[0]);
}

template <typename float_t>
void compute_and_store(const std::complex<float_t> &C, std::array<int, 2> rc,
                       const uint16_t maxit,
                       fractal_utils::fractal_map &map_age_u16,
                       fractal_utils::fractal_map *map_z) noexcept {
  using namespace libHybractal;
  std::complex<float_t> z{0, 0};
  int age = DECLARE_HYBRACTAL_SEQUENCE(
      HYBRACTAL_SEQUENCE_STR)::compute_age<float_t>(z, C, maxit);

  if (age < 0) {
    age = UINT16_MAX;
  }
  const int r = rc[0];
  const int c = rc[1];
  map_age_u16.at<uint16_t>(r, c) = static_cast<uint16_t>(age);
  if (map_z != nullptr) {
    map_z->at<std::complex<hybf_store_t>>(r, c).real(double(z.real()));
    map_z->at<std::complex<hybf_store_t>>(r, c).imag(double(z.imag()));
  }
}

template <typename float_t>
void compute_rest(const fractal_utils::center_wind<float_t> &wind_C,
                  const uint16_t maxit, fractal_utils::fractal_map &map_age_u16,
                  fractal_utils::fractal_map *map_z_nullable,
                  std::array<int, 2> rest_rc_start) noexcept {

  const std::complex<float_t> left_top{wind_C.left_top_corner()[0],
                                       wind_C.left_top_corner()[1]};
  const float_t r_unit = -wind_C.y_span / map_age_u16.rows;
  const float_t c_unit = wind_C.x_span / map_age_u16.cols;

#pragma omp parallel for schedule(dynamic)
  for (int r = 0; r < map_age_u16.rows; r++) {
    for (int c = rest_rc_start[1]; c < map_age_u16.cols; c++) {
      const float_t real = left_top.real() + c * c_unit;
      const float_t imag = left_top.imag() + r * r_unit;

      compute_and_store<float_t>({real, imag}, {r, c}, maxit, map_age_u16,
                                 map_z_nullable);
    }
  }

#pragma omp parallel for schedule(dynamic)
  for (int c = 0; c < rest_rc_start[1]; c++) {
    for (int r = rest_rc_start[0]; r < map_age_u16.rows; r++) {
      const float_t real = left_top.real() + c * c_unit;
      const float_t imag = left_top.imag() + r * r_unit;

      compute_and_store<float_t>({real, imag}, {r, c}, maxit, map_age_u16,
                                 map_z_nullable);
    }
  }
}

std::string
libHybractal::compute_frame_cuda(const fractal_utils::wind_base &wind_C,
                                 int precision, const uint16_t maxit,
                                 fractal_utils::fractal_map &map_age_u16,
                                 fractal_utils::fractal_map *map_z_nullable,
                                 cubractal_resource &gpu_rcs) noexcept {

  if (precision != 1 && precision != 2) {
    return fmt::format(
        "Invalid precision {} for gpu. Only 1 and 2 are supported.", precision);
  }

  if (!gpu_rcs.is_valid()) {
    return fmt::format("gpu_rcs is invalid");
  }

  if (map_age_u16.rows != gpu_rcs.rows() ||
      map_age_u16.cols != gpu_rcs.cols()) {
    return fmt::format("Size mismatch. Size of map_age_u16 is [{}, {}], but "
                       "size of gpu_rcs is [{}, {}]",
                       map_age_u16.rows, map_age_u16.cols, gpu_rcs.rows(),
                       gpu_rcs.cols());
  }

  if (map_z_nullable != nullptr) {
    if (map_age_u16.rows != map_z_nullable->rows ||
        map_age_u16.cols != map_z_nullable->cols) {
      return fmt::format("Size mismatch. Size of map_age_u16 is [{}, {}], but "
                         "size of map_z is [{}, {}]",
                         map_age_u16.rows, map_age_u16.cols,
                         map_z_nullable->rows, map_z_nullable->cols);
    }
  }

  constexpr int blk_rows = 8;
  constexpr int blk_cols = 8;

  const int row_num = map_age_u16.rows / blk_rows;
  const int col_num = map_age_u16.cols / blk_cols;

  const double r_unit = -wind_C.displayed_y_span() / map_age_u16.rows;
  const double c_unit = wind_C.displayed_x_span() / map_age_u16.cols;

  cuda_complex_t<double> left_top{wind_C.displayed_left_top_corner()[0],
                                  wind_C.displayed_left_top_corner()[1]};
  std::array<int, 2> size_rc{(int)map_age_u16.rows, (int)map_age_u16.cols};
  hipDoubleComplex *const gpu_ptr_z =
      (hipDoubleComplex *)((map_z_nullable != nullptr) ? (gpu_rcs.data_z_gpu())
                                                      : (nullptr));
  if (precision == 1) {
    compute_by_block<float>
        <<<dim3(col_num, row_num), dim3(blk_cols, blk_rows)>>>(
            size_rc, {left_top, r_unit, c_unit}, maxit, gpu_rcs.data_age_gpu(),
            gpu_ptr_z);
  } else {

    compute_by_block<double>
        <<<dim3(col_num, row_num), dim3(blk_cols, blk_rows)>>>(
            size_rc, {left_top, r_unit, c_unit}, maxit, gpu_rcs.data_age_gpu(),
            gpu_ptr_z);
  }

  hipError_t err;

  err = hipMemcpy(map_age_u16.data, gpu_rcs.data_age_gpu(),
                   map_age_u16.byte_count(),
                   hipMemcpyKind::hipMemcpyDeviceToHost);
  if (err != hipSuccess) {
    return fmt::format("hipMemcpy failed to copy map_age with error code {}",
                       err);
  }

  if (map_z_nullable != nullptr) {

    err = hipMemcpy(map_z_nullable->data, gpu_ptr_z,
                     map_z_nullable->byte_count(),
                     hipMemcpyKind::hipMemcpyDeviceToHost);
    if (err != hipSuccess) {
      return fmt::format("hipMemcpy failed to copy map_z with error code {}",
                         err);
    }
  }

  const int rest_row_start = map_age_u16.rows - row_num * blk_rows;
  const int rest_col_start = map_age_u16.cols - col_num * blk_cols;

  if (precision == 1) {
    compute_rest<float>(
        dynamic_cast<const fractal_utils::center_wind<float> &>(wind_C), maxit,
        map_age_u16, map_z_nullable, {rest_row_start, rest_col_start});
  } else {
    compute_rest<double>(
        dynamic_cast<const fractal_utils::center_wind<double> &>(wind_C), maxit,
        map_age_u16, map_z_nullable, {rest_row_start, rest_col_start});
  }

  return {};
}