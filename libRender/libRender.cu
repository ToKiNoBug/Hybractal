#include "hip/hip_runtime.h"
#ifndef HYBRACTAL_LIBRENDER_LIBRENDER_INTERNNAL_H
#define HYBRACTAL_LIBRENDER_LIBRENDER_INTERNNAL_H

#include "libRender.h"
#include <cmath>
#include <hip/hip_complex.h>
#include <hip/hip_runtime.h>

libHybractal::gpu_resource::gpu_resource(size_t _rows, size_t _cols)
    : m_rows(_rows), m_cols(_cols) {

  hipMalloc(&this->device_mat_age, _rows * _cols);
  hipMalloc(&this->device_mat_z, _rows * _cols);
  hipMalloc(&this->device_mat_u8c3, _rows * _cols);
}

libHybractal::gpu_resource::~gpu_resource() {
  if (device_mat_age != nullptr) {
    hipFree(device_mat_age);
  }
  if (device_mat_z != nullptr) {
    hipFree(device_mat_z);
  }
  if (device_mat_u8c3 != nullptr) {
    hipFree(device_mat_u8c3);
  }
}

__device__ uchar3 hsv2rgb(const float3 HSV) noexcept {
  const float H = HSV.x;
  const float S = HSV.y;
  const float V = HSV.z;
  const float C = V * S;

  const int H_i = H;

  const int H_mod_60 = H_i % 60;

  const float X = C * (1 - std::abs((H_i / 60) % 2 - 1));
  const float m = V - C;

  float3 results[6];

  results[0] = {C, X, 0};
  results[1] = {X, C, 0};
  results[2] = {0, C, X};
  results[3] = {0, X, C};
  results[4] = {X, 0, C};
  results[5] = {C, 0, X};

  float3 RGB_ = results[H_mod_60];

  RGB_.x += m;
  RGB_.y += m;
  RGB_.z += m;

  uchar3 ret;

  ret.x = RGB_.x * 255;
  ret.y = RGB_.y * 255;
  ret.z = RGB_.z * 255;

  return ret;
}

struct cplx {
  float norm;
  float arg;
};

__device__ cplx complex_convert(hipFloatComplex z) {
  cplx ret;
  ret.norm = sqrtf(z.x * z.x + z.y * z.y);

  ret.arg = std::atan2(z.x, z.y);
  return ret;
}

__device__ cplx cplx_cvt_normalize(hipFloatComplex z) {
  cplx ret = complex_convert(z);

  ret.norm /= 2;
  ret.arg = (ret.arg + M_PI) / (2 * M_PI);

  ret.arg = std::min(1.0f, std::max(0.0f, ret.arg));
  return ret;
}

__device__ float normalize_age_cos(uint16_t age, const float peroid) {
  return -0.5f * (std::cos(peroid / (2 * M_PI) * age) - 1);
}

__device__ float get_float3_value(float3 val, int idx) {
  float temp[3]{val.x, val.y, val.z};
  return temp[idx];
}

__device__ float3
map_value(float3 src, const libHybractal::hsv_render_option::hsv_range &range) {
  float3 hsv;

  hsv.x = (range.range_H[1] - range.range_H[0]) *
              get_float3_value(src, range.fv_mapping[0]) +
          range.range_H[0];

  hsv.y = (range.range_S[1] - range.range_S[0]) *
              get_float3_value(src, range.fv_mapping[1]) +
          range.range_S[0];

  hsv.z = (range.range_V[1] - range.range_V[0]) *
              get_float3_value(src, range.fv_mapping[2]) +
          range.range_V[0];
  return hsv;
}

__global__ void render_custom(const uint16_t *age_ptr,
                              const hipDoubleComplex *z_ptr, uchar3 *u8c3_ptr,
                              const libHybractal::hsv_render_option opt) {
  static_assert(sizeof(uchar3) == 3, "");

  const int gidx = blockIdx.x * blockDim.x + threadIdx.x;
  const uint16_t age = age_ptr[gidx];
  const hipFloatComplex z{(float)z_ptr[gidx].x, (float)z_ptr[gidx].y};

  const bool is_normal = (age < libHybractal::maxit_max);

  const libHybractal::hsv_render_option::hsv_range &range =
      (is_normal) ? opt.range_age_normal : opt.range_age_inf;

  const auto normalized = cplx_cvt_normalize(z);

  const float age_normalized = normalize_age_cos(age, range.age_peroid);

  float3 HSV =
      map_value({age_normalized, normalized.norm, normalized.arg}, range);

  u8c3_ptr[gidx] = hsv2rgb(HSV);

  // auto ret = range.map_value({age_normalized, normalized.norm,
  // normalized.arg});
}

#define handle_error(err)                                                      \
  if (err)                                                                     \
    abort();

__host__ void
libHybractal::render_hsv(const fractal_utils::fractal_map &mat_age,
                         const fractal_utils::fractal_map &mat_z,
                         fractal_utils::fractal_map &mat_u8c3,
                         const hsv_render_option &opt,
                         gpu_resource &rcs) noexcept {
  assert(rcs.ok());

  assert(rcs.rows() == mat_age.rows);
  assert(rcs.rows() == mat_z.rows);
  assert(rcs.rows() == mat_u8c3.rows);

  assert(rcs.cols() == mat_age.cols);
  assert(rcs.cols() == mat_z.cols);
  assert(rcs.cols() == mat_u8c3.cols);

  hipError_t err;

  err = hipMemcpy(rcs.mat_age_gpu(), mat_age.data, mat_age.byte_count(),
                   hipMemcpyKind::hipMemcpyHostToDevice);
  handle_error(err);

  err = hipMemcpy(rcs.mat_z_gpu(), mat_z.data, mat_z.byte_count(),
                   hipMemcpyKind::hipMemcpyHostToDevice);
  handle_error(err);

  err = hipMemset(rcs.mat_u8c3_gpu(), 0xFF, mat_u8c3.byte_count());
  handle_error(err);

  static_assert(sizeof(hipDoubleComplex) == sizeof(std::complex<double>), "");
  static_assert(sizeof(uchar3) == sizeof(fractal_utils::pixel_RGB), "");

  render_custom<<<mat_age.element_count(), 128>>>(
      rcs.mat_age_gpu(), (const hipDoubleComplex *)rcs.mat_z_gpu(),
      (uchar3 *)rcs.mat_u8c3_gpu(), opt);

  err = hipMemcpy(mat_u8c3.data, rcs.mat_u8c3_gpu(), mat_u8c3.byte_count(),
                   hipMemcpyKind::hipMemcpyDeviceToHost);
  handle_error(err);
}

#endif // HYBRACTAL_LIBRENDER_LIBRENDER_INTERNNAL_H