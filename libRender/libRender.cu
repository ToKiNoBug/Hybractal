#include "hip/hip_runtime.h"
/*
 Copyright © 2023  TokiNoBug
This file is part of Hybractal.

    Hybractal is free software: you can redistribute it and/or modify
    it under the terms of the GNU General Public License as published by
    the Free Software Foundation, either version 3 of the License, or
    (at your option) any later version.

    Hybractal is distributed in the hope that it will be useful,
    but WITHOUT ANY WARRANTY; without even the implied warranty of
    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
    GNU General Public License for more details.

    You should have received a copy of the GNU General Public License
    along with Hybractal.  If not, see <https://www.gnu.org/licenses/>.
*/

#ifndef HYBRACTAL_LIBRENDER_LIBRENDER_INTERNNAL_H
#define HYBRACTAL_LIBRENDER_LIBRENDER_INTERNNAL_H

#include "libRender.h"
#include <cmath>
#include <hip/hip_complex.h>
#include <hip/hip_runtime.h>
#include <iostream>

#define PRIVATE_HANDLE_ERROR_GPU_RCS(err_code)                                 \
  if (err_code) {                                                              \
    std::cerr << "hipMalloc failed. err_code = " << err_code << std::endl;    \
  }

libHybractal::gpu_resource::gpu_resource(size_t _rows, size_t _cols)
    : m_rows(_rows), m_cols(_cols) {
  hipError_t err_code;
  err_code =
      hipMalloc(&this->device_mat_age, _rows * _cols * sizeof(uint16_t));
  PRIVATE_HANDLE_ERROR_GPU_RCS(err_code);
  err_code = hipMalloc(&this->device_mat_z,
                        _rows * _cols *
                            sizeof(std::complex<libHybractal::hybf_store_t>));
  PRIVATE_HANDLE_ERROR_GPU_RCS(err_code);
  err_code =
      hipMalloc(&this->device_mat_u8c3, _rows * _cols * sizeof(uint8_t[3]));
  PRIVATE_HANDLE_ERROR_GPU_RCS(err_code);
}

libHybractal::gpu_resource::gpu_resource(gpu_resource &&another) {
  this->m_rows = another.m_rows;
  this->m_cols = another.m_cols;

  this->device_mat_age = another.device_mat_age;
  this->device_mat_u8c3 = another.device_mat_u8c3;
  this->device_mat_z = another.device_mat_z;

  another.device_mat_age = nullptr;
  another.device_mat_u8c3 = nullptr;
  another.device_mat_z = nullptr;
}

libHybractal::gpu_resource::~gpu_resource() {
  if (device_mat_age != nullptr) {
    hipFree(device_mat_age);
  }
  if (device_mat_z != nullptr) {
    hipFree(device_mat_z);
  }
  if (device_mat_u8c3 != nullptr) {
    hipFree(device_mat_u8c3);
  }
}

__device__ uchar3 hsv2rgb(const float3 HSV) noexcept {
  const float H = HSV.x;
  const float S = HSV.y;
  const float V = HSV.z;
  const float C = V * S;

  const int H_i = H;

  const int H_mod_60 = H_i / 60;

  const float X = C * (1 - std::abs((H_i / 60) % 2 - 1));
  const float m = V - C;

  float3 results[6];

  results[0] = {C, X, 0};
  results[1] = {X, C, 0};
  results[2] = {0, C, X};
  results[3] = {0, X, C};
  results[4] = {X, 0, C};
  results[5] = {C, 0, X};

  float3 RGB_ = results[H_mod_60];

  RGB_.x += m;
  RGB_.y += m;
  RGB_.z += m;

  uchar3 ret;

  ret.x = RGB_.x * 255;
  ret.y = RGB_.y * 255;
  ret.z = RGB_.z * 255;

  return ret;
}

struct cplx {
  float norm;
  float arg;
};

__device__ cplx complex_convert(hipFloatComplex z) {
  cplx ret;
  ret.norm = sqrtf(z.x * z.x + z.y * z.y);

  ret.arg = std::atan2(z.x, z.y);
  return ret;
}

__device__ cplx cplx_cvt_normalize(hipFloatComplex z) {
  cplx ret = complex_convert(z);

  ret.norm /= 2;
  ret.arg = (ret.arg + M_PI) / (2 * M_PI);

  ret.arg = std::min(1.0f, std::max(0.0f, ret.arg));
  return ret;
}

__device__ float normalize_age_cos(uint16_t age, const float peroid) {

  const float omega = 2 * M_PI / peroid;

  return 0.5f * (1 - std::cos(omega * age));
}

__device__ float get_float3_value(float3 val, int idx) {
  float temp[3]{val.x, val.y, val.z};
  return temp[idx];
}

__device__ float3
map_value(float3 src, const libHybractal::hsv_render_option::hsv_range &range) {
  float3 hsv;

  hsv.x = (range.range_H[1] - range.range_H[0]) *
              get_float3_value(src, range.fv_mapping[0]) +
          range.range_H[0];

  hsv.y = (range.range_S[1] - range.range_S[0]) *
              get_float3_value(src, range.fv_mapping[1]) +
          range.range_S[0];

  hsv.z = (range.range_V[1] - range.range_V[0]) *
              get_float3_value(src, range.fv_mapping[2]) +
          range.range_V[0];
  return hsv;
}

__global__ void render_custom(const uint16_t *age_ptr,
                              const hipDoubleComplex *z_ptr, uchar3 *u8c3_ptr,
                              const libHybractal::hsv_render_option opt) {
  static_assert(sizeof(uchar3) == 3, "");

  const int gidx = blockIdx.x * blockDim.x + threadIdx.x;
  const uint16_t age = age_ptr[gidx];
  const hipFloatComplex z{(float)z_ptr[gidx].x, (float)z_ptr[gidx].y};

  const bool is_normal = (age < libHybractal::maxit_max);

  const libHybractal::hsv_render_option::hsv_range &range =
      (is_normal) ? opt.range_age_normal : opt.range_age_inf;

  const auto normalized = cplx_cvt_normalize(z);

  const float age_normalized = normalize_age_cos(age, range.age_peroid);

  float3 HSV =
      map_value({age_normalized, normalized.norm, normalized.arg}, range);

  u8c3_ptr[gidx] = hsv2rgb(HSV);

  // auto ret = range.map_value({age_normalized, normalized.norm,
  // normalized.arg});
}

#define handle_error(err)                                                      \
  if (err) {                                                                   \
    std::cerr << "cuda error : " << err << std::endl;                          \
    abort();                                                                   \
  }

__host__ void
libHybractal::render_hsv(const fractal_utils::fractal_map &mat_age,
                         const fractal_utils::fractal_map &mat_z,
                         fractal_utils::fractal_map &mat_u8c3,
                         const hsv_render_option &opt,
                         gpu_resource &rcs) noexcept {
  assert(rcs.ok());

  assert(rcs.rows() == mat_age.rows);
  assert(rcs.rows() == mat_z.rows);
  assert(rcs.rows() == mat_u8c3.rows);

  assert(rcs.cols() == mat_age.cols);
  assert(rcs.cols() == mat_z.cols);
  assert(rcs.cols() == mat_u8c3.cols);

  hipError_t err;

  err = hipMemcpy(rcs.mat_age_gpu(), mat_age.data, mat_age.byte_count(),
                   hipMemcpyKind::hipMemcpyHostToDevice);
  handle_error(err);

  err = hipMemcpy(rcs.mat_z_gpu(), mat_z.data, mat_z.byte_count(),
                   hipMemcpyKind::hipMemcpyHostToDevice);
  handle_error(err);

  err = hipMemset(rcs.mat_u8c3_gpu(), 0xFF, mat_u8c3.byte_count());
  handle_error(err);

  static_assert(std::is_same_v<libHybractal::hybf_store_t, double>, "");

  static_assert(sizeof(hipDoubleComplex) ==
                    sizeof(std::complex<libHybractal::hybf_store_t>),
                "");
  static_assert(sizeof(uchar3) == sizeof(fractal_utils::pixel_RGB), "");

  const int blockdim = 64;

  render_custom<<<mat_age.element_count() / blockdim, blockdim>>>(
      rcs.mat_age_gpu(), (const hipDoubleComplex *)rcs.mat_z_gpu(),
      (uchar3 *)rcs.mat_u8c3_gpu(), opt);

  err = hipMemcpy(mat_u8c3.data, rcs.mat_u8c3_gpu(), mat_u8c3.byte_count(),
                   hipMemcpyKind::hipMemcpyDeviceToHost);
  if (err) {
    std::cerr << "mat_u8c3.data = " << mat_u8c3.data
              << ", rcs.mat_u8c3_gpu() = " << rcs.mat_u8c3_gpu() << std::endl;
  }

  handle_error(err);
}

#endif // HYBRACTAL_LIBRENDER_LIBRENDER_INTERNNAL_H